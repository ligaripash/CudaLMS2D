#include "hip/hip_runtime.h"

#include "LMS2D.h"

#include "hip/hip_runtime.h"

#include <iostream>
#include <fstream>
#include <Windows.h>
using namespace std;
#include <thrust/device_vector.h>
#include <sstream>



#define MAX_POINT_COUNT 2048


/*******************************************************************************************************/



LMS2D::LMS2D()
{
	initCUDA();
}


/*******************************************************************************************************/



LMS2D::~LMS2D()
{

	hipFree(mDeviceIntersectionPoints);
	hipFree(mDeviceMinBraceletPerIntersectionPoint);
	hipFree(mDeviceMinBraceletMidPintPerIntersectionPoint);
	hipFree(mDeviceInputLines);
	

	delete [] mHostInputLines;
	free(mHostMinBraceletReductionOutput);
	 

	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        cerr<<"hipDeviceReset failed!";
     
    }
}


/*******************************************************************************************************/


void LMS2D::initCUDA()
{
	hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
		cerr << "hipSetDevice failed!";
    }
}



/*******************************************************************************************************/

__device__ float calcYatXIntersection(int line_index, float* input_lines, float x)
{
	float m = input_lines[2*line_index];
	float n = input_lines[2*line_index + 1];

	float y = m * x + n;

	return y;
}




/*******************************************************************************************************/




__device__ void calculateInstersectionPoint(int col, int row, float* lines, float* intersection_p_x, float* intersection_p_y)
{
	
	float m1 = lines[2*col];

	float n1 = lines[2*col + 1];


	float m2 = lines[2*row];

	float n2 = lines[2*row + 1];

	if ( m1 == m2 ){
		// Handle degenrecies
		*intersection_p_x = FLT_MAX;

		*intersection_p_y = FLT_MAX;

		return ;
	}

	*intersection_p_x = (n2 - n1) / (m1 - m2);

	*intersection_p_y = m1 * (*intersection_p_x) + n1;

	
}




/*******************************************************************************************************/



__device__ int getIndex(int col, int row, int line_count)
{
	int index = 2 * (row * line_count + col - ( (row+1) * (row + 2) / 2 ));

	return index;
}



/*******************************************************************************************************/


__global__ void computeIntersectionPoints(float* input_lines,
										  int lines_count,
										  float*  mDeviceIntersectionPoints)
{
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row >= col) {
		return;
	}

	//now we have to calculate the instersection point of line #col width line #row

	float intersection_p_x;
	float intersection_p_y;

	calculateInstersectionPoint(col, row, input_lines, &intersection_p_x, &intersection_p_y);

	int index_for_x_coord_per_intersection_point = getIndex(col, row, lines_count);
	//int index_for_x_coord_per_intersection_point = 2;

	mDeviceIntersectionPoints[index_for_x_coord_per_intersection_point] = intersection_p_x;
	mDeviceIntersectionPoints[index_for_x_coord_per_intersection_point + 1] = intersection_p_y;


}


/*******************************************************************************************************/


inline __device__ void Comparator(
    float &keyA,
    float &keyB,
    unsigned int arrowDir
)
{
    float t;

    if ((keyA > keyB) == arrowDir)
    {
        t = keyA;
        keyA = keyB;
        keyB = t;
    }
}



/*******************************************************************************************************/


__device__ void bitonicSortSharedKernel(
    float *s_key,
    int tid,
    unsigned int arrayLength)
{

	int sortDir = 1;

//	#pragma unroll

    for (unsigned int size = 2; size < arrayLength; size <<= 1)
    {
        //Bitonic merge
        unsigned int dir = (tid & (size >> 1)) != 0;

        for (unsigned int stride = size >> 1; stride > 0; stride >>= 1)
        {
            __syncthreads();
            unsigned int pos = 2 * tid - (tid & (stride - 1));
            Comparator( s_key[pos +      0],  s_key[pos + stride],  dir     );
        }
    }

	
    //ddd == sortDir for the last bitonic merge step
    {
        for (unsigned int stride = arrayLength >> 1; stride > 0; stride >>= 1)
        {
            __syncthreads();
            unsigned int pos = 2 * tid - (tid & (stride - 1));
            Comparator(  s_key[pos +      0], s_key[pos + stride], sortDir);
            
        }
    }

    __syncthreads();
}



/*******************************************************************************************************/



__device__ void calculateMinBraceletSort(float local_y,
									 float intersection_p_y, 
									 int tid,
									 float* smem,			 
									 int line_count,					 
									 float* min_y_bracelet,
									 float* bracelet_mid_point)
{

	//First sort the values in shared memory
	bitonicSortSharedKernel(smem, tid, line_count);

	// Get the intersection_p_y location in the sorted sequence

	float f1 = smem[tid];
	//gil
	//float f2 = smem[tid + line_count / 2 - 1];
	float f2 = smem[tid + line_count / 2 ];

	if (f1 == intersection_p_y){

		if ((tid != 0) && smem[tid - 1] == intersection_p_y){
			return;
		}
		*min_y_bracelet = (abs(f2 - f1));
//		*bracelet_mid_point = (f1 + f2) / 2;
		*bracelet_mid_point = (f1 + f2) ;

	}
	if (f2 == intersection_p_y ){
		if ((tid != line_count - 1) && smem[tid + 1] == intersection_p_y){
			return;
		}

		*min_y_bracelet = (abs(f2 - f1));
//		*bracelet_mid_point = (f1 + f2) / 2;
		*bracelet_mid_point = (f1 + f2) ;

	}
	


}


/*******************************************************************************************************/


template <unsigned int blockSize>
__global__ void findGlobalMinimumBracelet(float* mDeviceMinBraceletPerIntersectionPoint, float* g_out)
{
	
	__shared__ float smem[2*MAX_POINT_COUNT];
	

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

	float val1 = mDeviceMinBraceletPerIntersectionPoint[i];
	float val2 = mDeviceMinBraceletPerIntersectionPoint[i + blockSize];
	float my_min = min(val1, val2);

    smem[tid] = my_min;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s=blockDim.x/2; s>32; s>>=1)
    {
        if (tid < s)
        {
            smem[tid] = my_min = min(my_min, smem[tid + s]);
        }

        __syncthreads();
    }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
        volatile float *sdata= smem ;

        if (blockSize >=  64)
        {
            sdata[tid] = my_min = min(my_min,sdata[tid + 32]);
        }

        if (blockSize >=  32)
        {
            sdata[tid] = my_min = min(my_min, sdata[tid + 16]);
        }

        if (blockSize >=  16)
        {
            sdata[tid] = my_min = min(my_min,sdata[tid +  8]);
        }

        if (blockSize >=   8)
        {
            sdata[tid] = my_min = min(my_min, sdata[tid +  4]);
        }

        if (blockSize >=   4)
        {
            sdata[tid] = my_min = min(my_min, sdata[tid +  2]);
        }

        if (blockSize >=   2)
        {
            sdata[tid] = my_min = min(my_min,sdata[tid +  1]);
        }
    }

    // write result for this block to global mem
    if (tid == 0) g_out[2*blockIdx.x] = smem[0];

	 __syncthreads();

	if (smem[0] == val1){
		g_out[2*blockIdx.x + 1] = i;
	}

	if (smem[0] == val2){
		g_out[2*blockIdx.x + 1] = i + blockSize;
	}

}


/*

We activate this kernel with one thread block per intersection point.
Currently all y intersction will be done in shared memory.

*/

__global__ void FindMinimalBraceletPerIntersectionPoint(float* input_lines,
														int line_count, 
														float*  mDeviceIntersectionPoints, 
														float*  mDeviceMinBraceletPerIntersectionPoint,
														float*  mDeviceMinBraceletMidPintPerIntersectionPoint)
{
	
	//if (blockIdx.x != 0){
	//	return;
	//}
	extern __shared__ float smem[];

	float intersection_x;
	float intersection_y;

	int tid = threadIdx.y;


	intersection_x = mDeviceIntersectionPoints[blockIdx.x * 2];
	intersection_y = mDeviceIntersectionPoints[blockIdx.x * 2 + 1];

	if (intersection_x == FLT_MAX){
		// No intersection point - parallel lines
		if (tid == 0){
			mDeviceMinBraceletPerIntersectionPoint[blockIdx.x] = FLT_MAX;
		}
		
		return;
	}

	
	__syncthreads();

	int offset = line_count >> 1;
	float local_y1 = calcYatXIntersection(tid, input_lines, intersection_x);
	float local_y2 = calcYatXIntersection(tid + offset, input_lines, intersection_x);


	smem[tid] = local_y1;	
	smem[tid + offset] = local_y2;	
	__syncthreads();

	//if (blockIdx.x == 0){
	//	mDeviceMinBraceletPerIntersectionPoint[tid+256] = smem[tid];
	//}

	float min_y_bracelet = FLT_MAX;
	float bracelet_mid_point = FLT_MAX;


	calculateMinBraceletSort(local_y1,
		intersection_y, 
		tid,
		smem,			 
		line_count,					 
		&min_y_bracelet,
		&bracelet_mid_point);


	if (min_y_bracelet != FLT_MAX){

		mDeviceMinBraceletPerIntersectionPoint[blockIdx.x] = min_y_bracelet;
		mDeviceMinBraceletMidPintPerIntersectionPoint[blockIdx.x] = bracelet_mid_point;
	}
		

}


/*******************************************************************************************************/


void LMS2D::findGlobalMinBracelet(float* x_coord, float* min_bracelet, float* y_coord)
{

   int BLOCK_SIZE = 512 ;//was 1024
   
   while ((this->mInputIntersectionPointCount / (2*BLOCK_SIZE) <= 1)){
	   BLOCK_SIZE /= 2;
   }

   dim3 dim_min_bracelet(BLOCK_SIZE, 1);
   dim3 dim_grid_min_bracelet(this->mInputIntersectionPointCount / (2*BLOCK_SIZE),1);

    
   switch (BLOCK_SIZE){
   case 512:
	   findGlobalMinimumBracelet<512><<<dim_grid_min_bracelet, dim_min_bracelet>>>(mDeviceMinBraceletPerIntersectionPoint, mDeviceMinBraceletReductionOutput);
	   break;

   case 256:
	   findGlobalMinimumBracelet<256><<<dim_grid_min_bracelet, dim_min_bracelet>>>(mDeviceMinBraceletPerIntersectionPoint, mDeviceMinBraceletReductionOutput);
	   break;

   case 128:
	   findGlobalMinimumBracelet<128><<<dim_grid_min_bracelet, dim_min_bracelet>>>(mDeviceMinBraceletPerIntersectionPoint, mDeviceMinBraceletReductionOutput);
	   break;

   case 64:
	   findGlobalMinimumBracelet<64><<<dim_grid_min_bracelet, dim_min_bracelet>>>(mDeviceMinBraceletPerIntersectionPoint, mDeviceMinBraceletReductionOutput);
	   break;

   case 32:
	   findGlobalMinimumBracelet<32><<<dim_grid_min_bracelet, dim_min_bracelet>>>(mDeviceMinBraceletPerIntersectionPoint, mDeviceMinBraceletReductionOutput);
	   break;



   }
   

   int item_count_in_output = mInputIntersectionPointCount / (BLOCK_SIZE * 2);

   int output_size = 2 * item_count_in_output;

   hipMemcpy(mHostMinBraceletReductionOutput, mDeviceMinBraceletReductionOutput,output_size * sizeof(float), hipMemcpyDeviceToHost);

   float current_min = FLT_MAX;
   int current_index = -1;

   for ( int i = 0; i < item_count_in_output; i += 2){
	   if (mHostMinBraceletReductionOutput[i] < current_min){
		   current_min = mHostMinBraceletReductionOutput[i];
		   current_index = mHostMinBraceletReductionOutput[i+1];
	   }
   }

	hipMemcpy(x_coord, mDeviceIntersectionPoints + 2 * current_index, sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(y_coord, mDeviceMinBraceletMidPintPerIntersectionPoint + current_index , sizeof(float), hipMemcpyDeviceToHost);

	*y_coord /= -2;
	*min_bracelet = current_min;

}



/*******************************************************************************************************/


void LMS2D::compute()
{




	// 2. compute the intersection points of each pair of lines.
	// x = n2 - n1 / m1 - m2;
	// y = m1 * x + n1;
	// 3. For each intersection point do:
	//   3.1. Find the relevant direction (either up or down)
	//   3.2. Write all y coordinates above (bellow) each intersection point to memory
	//   3.3. Find the median.
	//   3.4. write the bracelet value to memory.
	// 4. Compute the minimum bracelet value.
	// 5. The LMS line is the dual of (Xmin, Ymin + min_bracelet).


	// Invoke kernel
	
    //dim3 dimBlock(16, 16);
	dim3 dimBlock(8, 8);
    dim3 dimGrid(mInputPointsCount / dimBlock.x, mInputPointsCount / dimBlock.y);


	hipDeviceSynchronize();

#ifdef PERF
	LARGE_INTEGER freq;
	QueryPerformanceFrequency(&freq);


	LARGE_INTEGER start, end;
	QueryPerformanceCounter(&start);


	//First calculate all intersection points


	hipEvent_t cuda_start, cuda_stop, sstart, sstop, min_start, min_stop;
	float time_intersection_points, time_bracelet_per_itersection_point, time_minimum_bracelet;
	hipEventCreate(&cuda_start);
	hipEventCreate(&cuda_stop);
	hipEventCreate(&sstart);
	hipEventCreate(&sstop);
	hipEventCreate(&min_start);
	hipEventCreate(&min_stop);

	hipEventRecord( cuda_start, 0 );
#endif


	computeIntersectionPoints<<<dimGrid, dimBlock>>>(mDeviceInputLines,
													 mInputPointsCount,
													 mDeviceIntersectionPoints );

#ifdef PERF
	hipEventRecord( cuda_stop, 0 );
	hipEventSynchronize( cuda_stop );
	hipEventElapsedTime( &time_intersection_points, cuda_start, cuda_stop );
#endif

	dim3 dim_block_bracelet(1, mInputPointsCount / 2);

	dim3 dim_grid_bracelet(this->mInputIntersectionPointCount,1);

	
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);


#ifdef PERF
	hipEventRecord( sstart, 0 );
#endif

	hipError_t rc;
	FindMinimalBraceletPerIntersectionPoint<<<dim_grid_bracelet, dim_block_bracelet, mInputPointsCount * 4>>>(mDeviceInputLines, 
		mInputPointsCount, 
		mDeviceIntersectionPoints,
		mDeviceMinBraceletPerIntersectionPoint,
		mDeviceMinBraceletMidPintPerIntersectionPoint);


	rc = hipGetLastError();
#ifdef PERF
	hipEventRecord( sstop, 0 );
	hipEventSynchronize( sstop );
	hipEventElapsedTime( &time_bracelet_per_itersection_point, sstart, sstop );

    hipDeviceSynchronize();
	QueryPerformanceCounter(&end);


	hipEventRecord( min_start, 0 );

#endif

	findGlobalMinBracelet(&mLMSLineSlope, &mLMSMinBracelet, &mLMSLineIntercept);
	
#ifdef PERF	
	hipEventRecord( min_stop, 0 );
	hipEventSynchronize( min_stop );
	hipEventElapsedTime( &time_minimum_bracelet, min_start, min_stop );



	//double time = (double)(end.QuadPart - start.QuadPart) / 1000.0;

	cout << "Time intersection points computation = " << time_intersection_points   << std::endl;
	cout << "Time bracelet per intersection points computation = " << time_bracelet_per_itersection_point   << std::endl;
	cout << "Time minimum bracelet computation = " << time_minimum_bracelet   << std::endl;
	cout << "Total time = " <<  (double)(end.QuadPart - start.QuadPart) / (freq.QuadPart / 1000.0)   << std::endl;


	hipEventDestroy( cuda_start );
	hipEventDestroy( cuda_stop );
#endif	


}




/*******************************************************************************************************/



void LMS2D::compute(float* input_points, int input_points_count, float* lms_line_slope, float* lms_line_intercept, float* min_bracelet)
{
	mInputPointsCount = input_points_count;

	//transform input points to the lines dual.
	for (int i = 0; i < mInputPointsCount; i++){
		mHostInputLines[2*i] = input_points[2*i]; //x -> m
		mHostInputLines[2*i + 1] = -input_points[2*i + 1]; //y -> -n
	}

	// copy lines to device memory
	hipError_t cudaStatus = hipMemcpy(mDeviceInputLines, mHostInputLines, mInputPointsCount * 2 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cerr << "hipMemcpy failed!";
    }

	mInputIntersectionPointCount = mInputPointsCount * (mInputPointsCount - 1) / 2 ;

	this->compute();


	*lms_line_slope = mLMSLineSlope;
	*lms_line_intercept = mLMSLineIntercept;
	*min_bracelet = mLMSMinBracelet;
}



/*******************************************************************************************************/


void LMS2D::allocate()
{

	//allocate host memory

	mHostInputLines = new float[MAX_POINT_COUNT*2]; //2 floats per point

	// move input data to device

	
    hipError_t cudaStatus = hipMalloc((void**)&mDeviceInputLines, MAX_POINT_COUNT * 2 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }


	unsigned int max_intersection_point_count = MAX_POINT_COUNT * (MAX_POINT_COUNT - 1) / 2 ;


    cudaStatus = hipMalloc((void**)&mDeviceIntersectionPoints , 2 * max_intersection_point_count * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }
	
#if 0
	int total_y_coordinates_count = max_intersection_point_count * MAX_POINT_COUNT;

    cudaStatus = hipMalloc((void**)&mDeviceYcoordinatesPerVerticalLine, total_y_coordinates_count * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }
#endif
	
    cudaStatus = hipMalloc((void**)&mDeviceMinBraceletReductionOutput, MAX_POINT_COUNT * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }


    cudaStatus = hipMalloc((void**)&mDeviceMinBraceletPerIntersectionPoint, max_intersection_point_count * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }

    cudaStatus = hipMalloc((void**)&mDeviceMinBraceletMidPintPerIntersectionPoint, max_intersection_point_count * sizeof(float));
    if (cudaStatus != hipSuccess) {
        cerr << "hipMalloc failed!";
    }

	mHostMinBraceletReductionOutput = (float*)malloc(MAX_POINT_COUNT * sizeof(float));

}


